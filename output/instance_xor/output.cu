#include "hip/hip_runtime.h"
#include <fstream>
#include "output.hpp"
#include "modules/fully_connected.pkmlm/fully_connected.hpp"
#include "modules/sigmoid.pkmlm/sigmoid.hpp"

using l0_size=PKML::Dimension<2>;
using l0_1_size=PKML::Dimension<4>;
using l1_2_size=PKML::Dimension<4>;
using l2_3_size=PKML::Dimension<1>;
using l3_size=PKML::Dimension<1>;

using l0_t=FullyConnected<l0_size,l0_1_size,FullyConnected_params{.learning_rate=0.5,}>;
using l1_t=Sigmoid<l0_1_size,l1_2_size,Sigmoid_params{}>;
using l2_t=FullyConnected<l1_2_size,l2_3_size,FullyConnected_params{.learning_rate=0.5,}>;
using l3_t=Sigmoid<l2_3_size,l3_size,Sigmoid_params{}>;

PKML::float_t * buf0;
PKML::float_t * buf0_1;
PKML::float_t * buf1_2;
PKML::float_t * buf2_3;
PKML::float_t * buf3;

PKML::float_t * l0_alloc;
PKML::float_t * l1_alloc;
PKML::float_t * l2_alloc;
PKML::float_t * l3_alloc;

static constexpr PKML::float_t *& input_buffer = buf0;
static constexpr PKML::float_t *& output_buffer = buf3;

InstanceXor::Network::Network() {
hipMalloc((void **) &buf0, l0_size::element_product * sizeof(PKML::float_t));
hipMalloc((void **) &buf0_1, l0_1_size::element_product * sizeof(PKML::float_t));
hipMalloc((void **) &buf1_2, l1_2_size::element_product * sizeof(PKML::float_t));
hipMalloc((void **) &buf2_3, l2_3_size::element_product * sizeof(PKML::float_t));
hipMalloc((void **) &buf3, l3_size::element_product * sizeof(PKML::float_t));

if constexpr (l0_t::memory_requirement > 0) hipMalloc((void **) &l0_alloc, l0_t::memory_requirement * sizeof(PKML::float_t));
if constexpr (l1_t::memory_requirement > 0) hipMalloc((void **) &l1_alloc, l1_t::memory_requirement * sizeof(PKML::float_t));
if constexpr (l2_t::memory_requirement > 0) hipMalloc((void **) &l2_alloc, l2_t::memory_requirement * sizeof(PKML::float_t));
if constexpr (l3_t::memory_requirement > 0) hipMalloc((void **) &l3_alloc, l3_t::memory_requirement * sizeof(PKML::float_t));

l0_t::init(l0_alloc);
l1_t::init(l1_alloc);
l2_t::init(l2_alloc);
l3_t::init(l3_alloc);
}

InstanceXor::Network::~Network() {
hipFree(buf0);
hipFree(buf0_1);
hipFree(buf1_2);
hipFree(buf2_3);
hipFree(buf3);

if constexpr (l0_t::memory_requirement > 0) hipFree(l0_alloc);
if constexpr (l1_t::memory_requirement > 0) hipFree(l1_alloc);
if constexpr (l2_t::memory_requirement > 0) hipFree(l2_alloc);
if constexpr (l3_t::memory_requirement > 0) hipFree(l3_alloc);
}

template<std::size_t total_threads>
__device__ void forward_d(
const std::size_t thread_index,
PKML::float_t * const _buf0,
PKML::float_t * const _buf0_1,
PKML::float_t * const _buf1_2,
PKML::float_t * const _buf2_3,
PKML::float_t * const _buf3,
PKML::float_t * const _l0_alloc,
PKML::float_t * const _l1_alloc,
PKML::float_t * const _l2_alloc,
PKML::float_t * const _l3_alloc) {
PKML::thread_gate<total_threads, l1_2_size::element_product>(thread_index, [thread_index, _buf0, _buf0_1, _buf1_2, _buf2_3, _buf3, _l0_alloc, _l1_alloc, _l2_alloc, _l3_alloc]() {
_buf0_1[thread_index] = l0_t::forward_gated(thread_index, _buf0, _l0_alloc);
_buf1_2[thread_index] = l1_t::forward_ungated(_buf0_1[thread_index], _l1_alloc);
});
__syncthreads();
PKML::thread_gate<total_threads, l3_size::element_product>(thread_index, [thread_index, _buf0, _buf0_1, _buf1_2, _buf2_3, _buf3, _l0_alloc, _l1_alloc, _l2_alloc, _l3_alloc]() {
_buf2_3[thread_index] = l2_t::forward_gated(thread_index, _buf1_2, _l2_alloc);
_buf3[thread_index] = l3_t::forward_ungated(_buf2_3[thread_index], _l3_alloc);
});
__syncthreads();
}

template<std::size_t total_threads>
__device__ void propogate_d(
const std::size_t thread_index,
PKML::float_t * const _buf0,
PKML::float_t * const _buf0_1,
PKML::float_t * const _buf1_2,
PKML::float_t * const _buf2_3,
PKML::float_t * const _buf3,
PKML::float_t * const _l0_alloc,
PKML::float_t * const _l1_alloc,
PKML::float_t * const _l2_alloc,
PKML::float_t * const _l3_alloc,
const PKML::float_t * const correct
) {
forward_d<total_threads>(
thread_index,
_buf0,
_buf0_1,
_buf1_2,
_buf2_3,
_buf3,
_l0_alloc,
_l1_alloc,
_l2_alloc,
_l3_alloc);
__shared__ PKML::float_t intermediate_costs[total_threads];
PKML::float_t * intermediate_costs_ptr = intermediate_costs;
PKML::thread_gate<total_threads, l3_size::element_product>(thread_index, [thread_index, intermediate_costs_ptr, correct, _buf3]() {
intermediate_costs_ptr[thread_index] = PKML::Math::sub(_buf3[thread_index], correct[thread_index]);
});
PKML::thread_gate<total_threads, l3_size::element_product>(thread_index, [thread_index, intermediate_costs_ptr, _buf0, _buf0_1, _buf1_2, _buf2_3, _buf3, _l0_alloc, _l1_alloc, _l2_alloc, _l3_alloc]() {
PKML::float_t intermediate = intermediate_costs_ptr[thread_index];
intermediate = PKML::Math::mul(intermediate, l3_t::backward_ungated(_buf2_3[thread_index], _buf3[thread_index], _l3_alloc));
l2_t::backward_gated(thread_index, intermediate_costs_ptr, _buf1_2, intermediate, _l2_alloc);
});
__syncthreads();
PKML::thread_gate<total_threads, l1_2_size::element_product>(thread_index, [thread_index, intermediate_costs_ptr, _buf0, _buf0_1, _buf1_2, _buf2_3, _buf3, _l0_alloc, _l1_alloc, _l2_alloc, _l3_alloc]() {
PKML::float_t intermediate = intermediate_costs_ptr[thread_index];
intermediate = PKML::Math::mul(intermediate, l1_t::backward_ungated(_buf0_1[thread_index], _buf1_2[thread_index], _l1_alloc));
l0_t::backward_gated(thread_index, intermediate_costs_ptr, _buf0, intermediate, _l0_alloc);
});
__syncthreads();
}

template<std::size_t total_threads>
__global__ void forward_k(
PKML::float_t * const _buf0,
PKML::float_t * const _buf0_1,
PKML::float_t * const _buf1_2,
PKML::float_t * const _buf2_3,
PKML::float_t * const _buf3,
PKML::float_t * const _l0_alloc,
PKML::float_t * const _l1_alloc,
PKML::float_t * const _l2_alloc,
PKML::float_t * const _l3_alloc) {
const std::size_t thread_index = blockIdx.x * blockDim.x + threadIdx.x;
forward_d<total_threads>(
thread_index,
_buf0,
_buf0_1,
_buf1_2,
_buf2_3,
_buf3,
_l0_alloc,
_l1_alloc,
_l2_alloc,
_l3_alloc);
}

void InstanceXor::Network::forward() {
forward_k<1 * 4><<<1, 4>>>(
buf0,
buf0_1,
buf1_2,
buf2_3,
buf3,
l0_alloc,
l1_alloc,
l2_alloc,
l3_alloc);
}

template<std::size_t total_threads>
__global__ void train_k(
std::size_t iterations,
std::size_t mult,
PKML::float_t * const _buf0,
PKML::float_t * const _buf0_1,
PKML::float_t * const _buf1_2,
PKML::float_t * const _buf2_3,
PKML::float_t * const _buf3,
PKML::float_t * const _l0_alloc,
PKML::float_t * const _l1_alloc,
PKML::float_t * const _l2_alloc,
PKML::float_t * const _l3_alloc,
const InstanceXor::Dataset::TrainingSet * const dataset,
const std::size_t dataset_size
) {
const std::size_t thread_index = blockIdx.x * blockDim.x + threadIdx.x;
for (std::size_t i = 0; i < iterations; i++) {
const InstanceXor::Dataset::TrainingSet & training_set = dataset[(i * mult) % dataset_size];
PKML::thread_gate<total_threads, l0_size::element_product>(thread_index, [thread_index, training_set, _buf0]() {
_buf0[thread_index] = training_set.inputs[thread_index]; // possible optimization by moving _buf0 pointer
});
__syncthreads();
propogate_d<total_threads>(
thread_index,
_buf0,
_buf0_1,
_buf1_2,
_buf2_3,
_buf3,
_l0_alloc,
_l1_alloc,
_l2_alloc,
_l3_alloc,
training_set.outputs
);
}
}

void InstanceXor::Network::train(std::size_t iterations, std::size_t mult, InstanceXor::Dataset & dataset) {
train_k<1 * 4><<<1, 4>>>(
iterations,
mult,
buf0,
buf0_1,
buf1_2,
buf2_3,
buf3,
l0_alloc,
l1_alloc,
l2_alloc,
l3_alloc,
dataset._data,
dataset.size()
);
hipDeviceSynchronize();
}

template<std::size_t total_threads>
__global__ void evaluate_k(
PKML::float_t * const _buf0,
PKML::float_t * const _buf0_1,
PKML::float_t * const _buf1_2,
PKML::float_t * const _buf2_3,
PKML::float_t * const _buf3,
PKML::float_t * const _l0_alloc,
PKML::float_t * const _l1_alloc,
PKML::float_t * const _l2_alloc,
PKML::float_t * const _l3_alloc,
const InstanceXor::Dataset::TrainingSet * const dataset,
const std::size_t dataset_size,
PKML::float_t * cost_ptr
) {
const std::size_t thread_index = blockIdx.x * blockDim.x + threadIdx.x;
PKML::float_t cost = 0;
for (std::size_t i = 0; i < dataset_size; i++) {
const InstanceXor::Dataset::TrainingSet & training_set = dataset[i];
PKML::thread_gate<total_threads, l0_size::element_product>(thread_index, [thread_index, training_set, _buf0]() {
_buf0[thread_index] = training_set.inputs[thread_index]; // possible optimization by moving _buf0 pointer
});
__syncthreads();
forward_d<total_threads>(
thread_index,
_buf0,
_buf0_1,
_buf1_2,
_buf2_3,
_buf3,
_l0_alloc,
_l1_alloc,
_l2_alloc,
_l3_alloc);
PKML::thread_gate<total_threads, l2_3_size::element_product>(thread_index, [thread_index, &cost, dataset_size, training_set, _buf3]() {
cost += (training_set.outputs[thread_index] - _buf3[thread_index]) / dataset_size;
});
}
*cost_ptr = cost;
}

PKML::float_t InstanceXor::Network::evaluate(InstanceXor::Dataset & dataset) {
PKML::float_t * dev_cost;
hipMalloc((void **) &dev_cost, sizeof(PKML::float_t));
evaluate_k<1 * 4><<<1, 4>>>(
buf0,
buf0_1,
buf1_2,
buf2_3,
buf3,
l0_alloc,
l1_alloc,
l2_alloc,
l3_alloc,
dataset._data,
dataset.size(),
dev_cost
);
PKML::float_t cost;
hipMemcpy(&cost, dev_cost, sizeof(float), hipMemcpyDeviceToHost);
return cost;
}

void InstanceXor::Network::save(const char * path) {
std::ofstream fs(path);
if constexpr (l0_t::memory_requirement != 0) {
float * temp = new float[l0_t::memory_requirement];
hipMemcpy(temp, l0_alloc, l0_t::memory_requirement * sizeof(float), hipMemcpyDeviceToHost);
fs.write((const char *) temp, l0_t::memory_requirement * sizeof(float));
delete[] temp;
}
if constexpr (l1_t::memory_requirement != 0) {
float * temp = new float[l1_t::memory_requirement];
hipMemcpy(temp, l1_alloc, l1_t::memory_requirement * sizeof(float), hipMemcpyDeviceToHost);
fs.write((const char *) temp, l1_t::memory_requirement * sizeof(float));
delete[] temp;
}
if constexpr (l2_t::memory_requirement != 0) {
float * temp = new float[l2_t::memory_requirement];
hipMemcpy(temp, l2_alloc, l2_t::memory_requirement * sizeof(float), hipMemcpyDeviceToHost);
fs.write((const char *) temp, l2_t::memory_requirement * sizeof(float));
delete[] temp;
}
if constexpr (l3_t::memory_requirement != 0) {
float * temp = new float[l3_t::memory_requirement];
hipMemcpy(temp, l3_alloc, l3_t::memory_requirement * sizeof(float), hipMemcpyDeviceToHost);
fs.write((const char *) temp, l3_t::memory_requirement * sizeof(float));
delete[] temp;
}
}

void InstanceXor::Network::load(const char * path) {
std::ifstream fs(path);
if constexpr (l0_t::memory_requirement != 0) {
float * temp = new float[l0_t::memory_requirement];
fs.read((char *) temp, l0_t::memory_requirement * sizeof(float));
hipMemcpy(l0_alloc, temp, l0_t::memory_requirement * sizeof(float), hipMemcpyHostToDevice);
delete[] temp;
}
if constexpr (l1_t::memory_requirement != 0) {
float * temp = new float[l1_t::memory_requirement];
fs.read((char *) temp, l1_t::memory_requirement * sizeof(float));
hipMemcpy(l1_alloc, temp, l1_t::memory_requirement * sizeof(float), hipMemcpyHostToDevice);
delete[] temp;
}
if constexpr (l2_t::memory_requirement != 0) {
float * temp = new float[l2_t::memory_requirement];
fs.read((char *) temp, l2_t::memory_requirement * sizeof(float));
hipMemcpy(l2_alloc, temp, l2_t::memory_requirement * sizeof(float), hipMemcpyHostToDevice);
delete[] temp;
}
if constexpr (l3_t::memory_requirement != 0) {
float * temp = new float[l3_t::memory_requirement];
fs.read((char *) temp, l3_t::memory_requirement * sizeof(float));
hipMemcpy(l3_alloc, temp, l3_t::memory_requirement * sizeof(float), hipMemcpyHostToDevice);
delete[] temp;
}
}

void InstanceXor::Network::copy_outputs(PKML::float_t * dst) {
hipMemcpy(dst, output_buffer, l3_size::element_product * sizeof(PKML::float_t), hipMemcpyDeviceToHost);
}

void InstanceXor::Network::copy_inputs(PKML::float_t * src) {
hipMemcpy(input_buffer, src, l0_size::element_product * sizeof(PKML::float_t), hipMemcpyHostToDevice);
}

InstanceXor::Dataset::Dataset(): _size(0), _capacity(1) {
if (hipMalloc((void **) &_data, _capacity * sizeof(TrainingSet)) == hipErrorOutOfMemory) throw std::bad_alloc();
}

InstanceXor::Dataset::~Dataset() {
hipFree(_data);
}

void InstanceXor::Dataset::push_back(const TrainingSet & value) {
if (_size == _capacity) {
_capacity *= 2;
TrainingSet * temp_mem;
if (hipMalloc((void **) &temp_mem, _capacity * sizeof(TrainingSet)) == hipErrorOutOfMemory) throw std::bad_alloc();
hipMemcpy(temp_mem, _data, _size * sizeof(TrainingSet), hipMemcpyDeviceToDevice);
hipFree(_data);
_data = temp_mem;
}
hipMemcpy(&_data[_size++], &value, sizeof(TrainingSet), hipMemcpyHostToDevice);
}

void InstanceXor::Dataset::pull_set(std::size_t index, TrainingSet & set) const {
hipMemcpy(&set, &_data[index], sizeof(TrainingSet), hipMemcpyDeviceToHost);
}

